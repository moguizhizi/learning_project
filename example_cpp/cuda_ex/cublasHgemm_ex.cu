#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    const int m = 4, n = 3, k = 2;
    const half alpha = __float2half(1.0f), beta = __float2half(0.0f);

    half h_A[m * k], h_B[k * n], h_C[m * n];
    for (int i = 0; i < m * k; ++i)
        h_A[i] = __float2half(i + 1.f);
    for (int i = 0; i < k * n; ++i)
        h_B[i] = __float2half(i + 1.f);

    half *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(half) * m * k);
    hipMalloc(&d_B, sizeof(half) * k * n);
    hipMalloc(&d_C, sizeof(half) * m * n);
    hipMemcpy(d_A, h_A, sizeof(half) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(half) * k * n, hipMemcpyHostToDevice);

    hipblasHandle_t h;
    hipblasCreate(&h);
    hipblasHgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_B, k, &beta, d_C, n);
    hipMemcpy(h_C, d_C, sizeof(half) * m * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j)
            printf("%6.2f ", __half2float(h_C[i * n + j]));
        printf("\n");
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(h);
    return 0;
}