#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    const int m = 4;
    const int n = 3;
    const int k = 2;

    // 主机内存
    float h_A[m * k] = {1, 2, 3, 4, 5, 6, 7, 8}; // m×k
    float h_B[k * n] = {1, 2, 3, 4, 5, 6};       // k×n
    float h_C[m * n] = {0};                      // m×n

    // 设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(float) * m * k);
    hipMalloc(&d_B, sizeof(float) * k * n);
    hipMalloc(&d_C, sizeof(float) * m * n);

    hipMemcpy(d_A, h_A, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * k * n, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, sizeof(float) * m * n, hipMemcpyHostToDevice);

    // CUBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 参数： C = 1·A·B + 0·C  → 纯乘法
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // 行主序 → cublas 用列主序，所以把 A,B 视为转置
    // 调用 cublasSgemm:  C = α·op(A)·op(B) + β·C
    hipblasSgemm(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N, // 不转置
                n,
                m,
                k, // 列主序维度：m,n,k
                &alpha,
                d_B,
                n, // B 是 k×n，leading dim = n
                d_A,
                k, // A 是 m×k，leading dim = k
                &beta,
                d_C,
                n + 1); // C 是 m×n，leading dim = n

    hipMemcpy(h_C, d_C, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    // 打印结果
    printf("Result C (m×n):\n");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j)
            printf("%6.1f ", h_C[i * n + j]);
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    return 0;
}