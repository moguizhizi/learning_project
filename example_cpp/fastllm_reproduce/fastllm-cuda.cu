#include "hip/hip_runtime.h"
#include "fastllm-cuda.cuh"

std::map<int, std::vector<CudaMemoryBuffer>> cudaBuffersMap;
std::map<int, int> cudaBuffersMinId;
std::map<int, size_t> noBusyCnt;
std::map<int, std::vector<CudaMemoryBuffer>> bigBuffersMap;

#define checkCudaErrors(message, val) showError(val, message, __FILE__, __LINE__)

void showError(hipError_t result, char const *const message, const char *const file, int const line) {
    if (hipSuccess != result) {
        printf("%s\n  CUDA error = %d, %s at %s:%d\n  '%s'\n", message, result, hipGetErrorName(result), file, line, hipGetErrorString(result));
    }
}

void *FastllmCudaMalloc(size_t size) {
    int id = -1;
    hipError_t state = hipGetDevice(&id);
    checkCudaErrors("Error: CUDA error when find device!", state);

    if (size > 1024 * 1024) {
        auto &bigBuffers = bigBuffersMap[id];
        int setId = -1;
        for (int i = 0; i < bigBuffers.size(); i++) {
            if (bigBuffers[i].size >= size && !bigBuffers[i].busy && (bigBuffers[i].size - size < 1024 * 1024)) {
                if (setId == -1 || bigBuffers[setId].size > bigBuffers[i].size) {
                    setId = i;
                }
            }
        }

        if (setId != -1) {
            bigBuffers[setId].busy = true;
            return bigBuffers[setId].data;
        }

        void *ret;

        hipError_t state = hipMalloc(&ret, size);
        if (state != hipSuccess) {
            printf("Error: CUDA error when allocating %lu MB memory! maybe there's no enough memory left on device.", size >> 20);
            checkCudaErrors("", state);
            return nullptr;
        }
        bigBuffers.push_back(CudaMemoryBuffer(ret, size, true));
        return ret;
    }

    auto &cudaBuffers = cudaBuffersMap[id];
    for (int i = cudaBuffersMinId[id]; i < cudaBuffers.size(); i++) {
        if (cudaBuffers[i].size > size && !cudaBuffers[i].busy) {
            cudaBuffers[i].busy = true;
            noBusyCnt[id] -= cudaBuffers[i].size;
            while (cudaBuffersMinId[id] < cudaBuffers.size() && cudaBuffers[cudaBuffersMinId[id]].busy) {
                cudaBuffersMinId[id]++;
            }
            return cudaBuffers[i].data;
        }
    }

    void *ret;

    state = hipMalloc(&ret, size);
    if (state != hipSuccess) {
        printf("Error: CUDA error when allocating %lu KB memory! maybe there's no enough memory left on device.", size >> 10);
        checkCudaErrors("", state);
        return nullptr;
    }
    cudaBuffers.push_back(CudaMemoryBuffer(ret, size, true));
    return ret;
}

void FastllmCudaFree(void *ret) {
    if (ret == nullptr) {
        return;
    }
    if (cudaBuffersMap.empty()) {
        return;
    }

    hipError_t state = hipSuccess;

    for (auto &it : cudaBuffersMap) {
        if (noBusyCnt[it.first] > 1024 * 1024 * 1024) {
            auto &cudaBuffers = it.second;
            std::vector<CudaMemoryBuffer> temp;
            for (int i = 0; i < cudaBuffers.size(); i++) {
                if (!cudaBuffers[i].busy) {
                    state = hipSetDevice(it.first);
                    state = hipFree(cudaBuffers[i].data);
                    if (state != hipSuccess) {
                        printf("Error: CUDA error when release memory on device %d!", it.first);
                    }
                    checkCudaErrors("", state);
                } else {
                    temp.push_back(cudaBuffers[i]);
                }
            }
            cudaBuffers.clear();
            it.second = temp;
            noBusyCnt[it.first] = 0;
        }
    }

    for (auto &it : cudaBuffersMap) {
        auto &cudaBuffers = it.second;
        for (int i = 0; i < cudaBuffers.size(); i++) {
            if (cudaBuffers[i].data == ret) {
                noBusyCnt[it.first] += cudaBuffers[i].size;
                cudaBuffers[i].busy = false;
                cudaBuffersMinId[it.first] = std::min(cudaBuffersMinId[it.first], i);
                return;
            }
        }
        auto &bigBuffers = bigBuffersMap[it.first];
        for (int i = 0; i < bigBuffers.size(); i++) {
            if (bigBuffers[i].data == ret) {
                bigBuffers[i].busy = false;
                return;
            }
        }
    }

    state = hipFree(ret);
    checkCudaErrors("CUDA error when release memory!", state);
}

void FastllmCudaCopyFromHostToDevice(void *dst, void *src, size_t size) {
    hipError_t state = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    checkCudaErrors("Error: CUDA error when copy from memory to GPU!", state);
}

void FastllmCudaCopyFromDeviceToHost(void *dst, void *src, size_t size) {
    hipError_t state = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    checkCudaErrors("Error: CUDA error when copy from GPU to memory!", state);
}

void FastllmCudaCopyFromDeviceToDevice(void *dst, void *src, size_t size) {
    hipError_t state = hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
    checkCudaErrors("Error: CUDA error when copy on GPU!", state);
}

void FastllmCudaMemcpyBetweenDevices(int dstId, void *dst, int srcId, void *src, size_t size) {
    int canAccess = 0;
    hipError_t state = hipDeviceCanAccessPeer(&canAccess, dstId, srcId);
    if (state == hipSuccess && canAccess) {
        hipMemcpyPeer(dst, dstId, src, srcId, size);
    } else {
        uint8_t *cpudata = new uint8_t[size];
        hipSetDevice(srcId);
        hipMemcpy(cpudata, src, size, hipMemcpyDeviceToHost);
        hipSetDevice(dstId);
        hipMemcpy(dst, cpudata, size, hipMemcpyHostToDevice);
        delete[] cpudata;
    }
    checkCudaErrors("Error: CUDA error when copy Between GPUs!", state);
    DeviceSync();
}

void FastllmCudaSetDevice(int gpu_id) { hipSetDevice(gpu_id); }

int FastllmCudaGetDevice() {
    int id = -1;
    hipGetDevice(&id);
    return id;
}

void DeviceSync() {
    // hipDeviceSynchronize();
}

void FastllmCudaClearBigBuffer() {
    int id = -1;
    hipGetDevice(&id);

    if (bigBuffersMap.empty()) {
        return;
    }

    for (auto &it : bigBuffersMap) {
        auto &bigBuffers = it.second;

        long long littleMemSum = 0;
        long long littleMemSumLimit = 300 * 1024 * 1024; // 留一小部分复用
        std::set<int> limitedID;
        std::vector<std::pair<size_t, int>> idle_size;
        for (int i = 0; i < bigBuffers.size(); i++) {
            if (!bigBuffers[i].busy) {
                idle_size.push_back(std::make_pair(bigBuffers[i].size, i));
            }
        }

        std::sort(idle_size.begin(), idle_size.end());
        for (int i = 0; i < idle_size.size(); i++) {
            littleMemSum += idle_size[i].first;

            if (littleMemSum > littleMemSumLimit) {
                break;
            } else {
                limitedID.insert(idle_size[i].second);
            }
        }

        std::vector<CudaMemoryBuffer> temp;
        for (int i = 0; i < bigBuffers.size(); i++) {
            if (!bigBuffers[i].busy && limitedID.find(i) == limitedID.end()) {
                hipError_t state = hipSuccess;
                hipSetDevice(it.first);
                state = hipFree(bigBuffers[i].data);
                if (hipSuccess != state)
                    printf("Error: CUDA error when release memory on device %d!", it.first);
                checkCudaErrors("", state);
            } else {
                temp.push_back(bigBuffers[i]);
            }
        }

        bigBuffers.clear();
        bigBuffers = temp;
    }

    hipSetDevice(id);
}

void FastllmCudaMemcpy2DDeviceToDevice(void *dst, size_t dpitch, const void *src, size_t spitch, size_t width, size_t height) {
    hipMemcpy2D(dst, dpitch, src, spitch, width, height, hipMemcpyDeviceToDevice);
    DeviceSync();
}

void *FastllmCudaDirectMalloc(size_t size) {
    void *ret;
    hipError_t state = hipMalloc(&ret, size);
    if (hipSuccess != state) {
        printf("Error: CUDA error when allocating %lu kB memory! maybe there's no enough memory left on device.", size >> 10);
        checkCudaErrors("", state);
        return nullptr;
    }
    return ret;
}

void FastllmCudaMemset0(void *ret, size_t size) { hipMemset(ret, 0, size); }
